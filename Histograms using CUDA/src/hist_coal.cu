#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime_api.h"

__global__ void hist_float_kernel(float *buffer, int *counts, int numObs, int numBins, float binWidth, float rangeStart, float rangeEnd)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < numObs;
		 i += blockDim.x * gridDim.x)
	{
		if (buffer[i] < rangeEnd && buffer[i] >= rangeStart)
		{
			int bin = floor((buffer[i] - rangeStart) / binWidth);
			atomicAdd(&counts[bin], 1);
		}
	}
}

int main(int argc, char *argv[])
{
	//printf("Please provide the number of observations as the last (7th) argument");
	FILE *inputFile = fopen(argv[1], "r");

	int numBins = atoi(argv[2]);
	float rangeStart = atof(argv[3]);
	float rangeEnd = atof(argv[4]);
	int gridDim = atoi(argv[5]);
	int blockDim = atoi(argv[6]);

	//Check if long is enough
	long numObs = atol(argv[7]);

	float *buffer = (float *)malloc(sizeof(float) * numObs);
	size_t count = fread(buffer, sizeof(float), numObs, inputFile);

	float *d_buffer;
	hipMalloc((void **)&d_buffer, sizeof(float) * numObs);
	hipMemcpy(d_buffer, buffer, sizeof(float) * numObs, hipMemcpyHostToDevice);

	int *d_counts;
	hipMalloc((void **)&d_counts, sizeof(int) * numBins);
	hipMemset(d_counts, 0, sizeof(int) * numBins);

	float binWidth = (rangeEnd - rangeStart) / numBins;

	hipEvent_t tick, tock;
	checkCudaErrors(hipEventCreate(&tick));
	checkCudaErrors(hipEventCreate(&tock));

	checkCudaErrors(hipEventRecord(tick, 0));
	hist_float_kernel<<<gridDim, blockDim>>>(d_buffer, d_counts, numObs, numBins, binWidth, rangeStart, rangeEnd);
	checkCudaErrors(hipEventRecord(tock, 0));
	checkCudaErrors(hipEventSynchronize(tock));
	
	float time;
	checkCudaErrors(hipEventElapsedTime(&time, tick, tock));

	int *counts = (int *)malloc(sizeof(int) * numBins);
	hipMemcpy(counts, d_counts, sizeof(int) * numBins, hipMemcpyDeviceToHost);

	int num = 1;
	for (float i = rangeStart; i < rangeEnd; i += binWidth)
	{
		printf("Bin no. %d: [%f - %f). Count= %d \n", num, i, i + binWidth, counts[num - 1]);
		num++;
	}

	printf( "Time taken= %f ms\n", time);
	free(buffer);
	free(counts);

	hipEventDestroy(tick);
	hipEventDestroy(tock);
	hipFree(d_counts);
	hipFree(d_buffer);
}