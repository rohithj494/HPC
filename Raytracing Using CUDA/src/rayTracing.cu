#include "hip/hip_runtime.h"
extern "C"
{
#include "bitmap.h"
}

#include "hip/hip_runtime_api.h"
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>

__device__ float vecMod(float *vec)
{
	return sqrtf(pow(vec[0], 2) + pow(vec[1], 2) + pow(vec[2], 2));
}

__device__ float vecDot(float *v1, float *v2)
{
	float dot = (v1[0] * v2[0]) + (v1[1] * v2[1]) + (v1[2] * v2[2]);
	return dot;
}

__global__ void rayTrace(RgbQuad *data, int n)
{

	int Wmax = 10;
	float L[3] = {4, 4, -1};
	float C[3] = {0, 12, 0};
	int R = 6;
	float delX = (2 * Wmax) / (float)n;
	float delZ = delX;

	for (int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < n;
		 i += (gridDim.x * blockDim.x))
	{
		for (int j = (blockIdx.y * blockDim.y) + threadIdx.y; j < n;
			 j += (gridDim.y * blockDim.y))
		{

			float b;
			float Wx = (i * delX) - Wmax;
			float Wz = (j * delZ) - Wmax;

			float D = pow(R, 2) - pow((Wx - C[0]), 2) - pow((Wz - C[2]), 2);
			if (D < 0.0)
				b = 0.0;
			else
			{
				//float Iy = C[1] - sqrtf(D);
				float I[3] = {Wx, (C[1] - sqrtf(D)), Wz};
				//float tempN[3] = {I[0] - C[0], I[1] - C[1], I[2] - C[2]};
				float N[3] = {(I[0] - C[0]), (I[1] - C[1]), (I[2] - C[2])};
				float mod = vecMod(N);
				N[0]= N[0]/mod; N[1]=N[1]/mod; N[2]=N[2]/mod;
				//N[3] = {N[0]/ mod, N[1] / mod, N[2] / mod};
				//float tempS[3] = {L[0] - I[0], L[1] - I[1], L[2] - I[2]};
				float S[3] = {(L[0] - I[0]) , (L[1] - I[1]), (L[2] - I[2]) };
				mod = vecMod(S);
				S[0]= S[0]/mod; S[1]=S[1]/mod; S[2]=S[2]/mod;
				//float S[3] = {(L[0] - I[0]) / mod, (L[1] - I[1]) / mod, (L[2] - I[2]) / mod};

				float dot = vecDot(S, N);
				if (dot <= 0)
					b = 0.0;
				else
				{
					b = dot;
					RgbQuad *pix = &data[i * n + j];
					pix->red = (int)(b * 255);
					pix->green = (int)(b * 255);
					pix->blue = (int)(b * 255);
				}
			}
		}
	}
}

int main(int argc, char *argv[])
{
	hipEvent_t tick, tock;
	hipEventCreate(&tick);
	hipEventCreate(&tock);
	int gridX, gridY, blockX, blockY;
	gridX=atoi(argv[2]); gridY=atoi(argv[3]); blockX=atoi(argv[4]); blockY=atoi(argv[5]);
	int n = atoi(argv[1]);
	RgbQuad *data = (RgbQuad *)malloc(n * n * sizeof(RgbQuad));

	RgbQuad *d_img;
	hipMalloc((void **)&d_img, n * n * sizeof(RgbQuad));
	hipMemcpy(d_img, data, n * n * sizeof(RgbQuad), hipMemcpyHostToDevice);

	dim3 blockDim(blockX, blockY, 1);
	dim3 gridDim(gridX, gridY, 1);
	
	hipEventRecord(tick, 0);
	rayTrace<<<gridDim, blockDim>>>(d_img, n);
	hipEventRecord(tock, 0);

	hipEventSynchronize(tock);

	float time;
	hipEventElapsedTime(&time, tick, tock);

	printf("Time taken= %f ms", time);
	hipMemcpy(data, d_img, n * n * sizeof(RgbQuad), hipMemcpyDeviceToHost);
	save_bitmap(data, n, n, "demo.bmp");

	free(data);
	hipEventDestroy(tick);
	hipEventDestroy(tock);
	hipFree(d_img);
}
