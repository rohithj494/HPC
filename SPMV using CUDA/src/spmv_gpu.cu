
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

typedef struct
{
	int *data;
	int *transposed;
	int *col_index;
	int *transposed_index;
	size_t m;
	size_t n;
	size_t row_len;
	size_t data_len;
} EllMatrix;


//Kernel function for spmv
__global__ void ell_spmv(int *transposed, int *transposed_index, int *vectorx, int *vectory, int n, int cols){
    
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x)
   {
    int sum = 0;
    for (int j = 0; j < cols; j++)
    {
        sum += transposed[j * n + i] * vectorx[transposed_index[j * n + i]];
    }
    vectory[i] += sum;
   }
}


int main(int argc, char *argv[])
{

	if (argc < 5){
		printf ("Too few arguments. Exiting.\n");
		exit(0);
	}

	int n = atoi(argv[1]);
	float p = atof(argv[2]);
    int gridDim= atoi(argv[3]);
    int blockDim= atoi(argv[4]);

    //Reading input data
	char Abuf[50];
	char xbuf[50];
	char ybuf[50];

	snprintf(Abuf, 50, "A_%d.dat", n);
	snprintf(xbuf, 50, "x_%d.dat", n);
	snprintf(ybuf, 50, "y_%d.dat", n);

	FILE *Afile = fopen(Abuf, "rb");
	FILE *xfile = fopen(xbuf, "rb");
	FILE *yfile = fopen(ybuf , "rb");

	int32_t *fullMatrix = (int32_t *)malloc(n * n * sizeof(int32_t));
	int32_t *vectorx = (int32_t *)malloc(n * sizeof(int32_t));
	int32_t *vectory = (int32_t *)malloc(n * sizeof(int32_t));

	size_t count = fread(fullMatrix, sizeof(int32_t), n * n, Afile);
	count= fread(vectorx, sizeof(int32_t), n, xfile);
	count= fread(vectory, sizeof(int32_t), n, yfile);

	//Setting the sparse matrix
	EllMatrix *sparseStruct = (EllMatrix *)malloc(sizeof(EllMatrix));
	sparseStruct->m = n;
	sparseStruct->n = n;
	int maxRowLen;

	//Finding max row length
	for (int i = 0; i < n; i++)
	{
		maxRowLen = 0;
		for (int j = 0; j < n; j++)
		{
			if (fullMatrix[i * n + j] != 0)
			{
				maxRowLen++;
			}
		}
		if (maxRowLen >= sparseStruct->row_len)
		{
			sparseStruct->row_len = maxRowLen;
		}
	}

	//allocating space for eLL matrix
	sparseStruct->data_len = sparseStruct->row_len * n;
	sparseStruct->data = (int *)malloc(sizeof(int) * sparseStruct->data_len);
	sparseStruct->col_index = (int *)malloc(sizeof(int) * sparseStruct->data_len);
	sparseStruct->transposed = (int *)malloc(sizeof(int) * sparseStruct->data_len);
	sparseStruct->transposed_index= (int *)malloc(sizeof(int) * sparseStruct->data_len);

	//Setting up the ELL matrix
	int k = 0;
	for (int i = 0; i < n; i++)
	{
		int rowContent = 0;
		for (int j = 0; j < n; j++)
		{
			if (fullMatrix[(i * n) + j] != 0)
			{
				sparseStruct->data[k] = fullMatrix[i * n + j];
				sparseStruct->col_index[k] = j;
				++k;
				++rowContent;
			}
		}

		//Padding with zeros
		while (rowContent < sparseStruct->row_len)
		{
			sparseStruct->col_index[k] = 0;
			sparseStruct->data[k] = 0;
			++k;
			++rowContent;
		}
	}

	//Transposing
	k = sparseStruct->row_len;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < sparseStruct->row_len; j++)
		{
			sparseStruct->transposed[j * n + i] = sparseStruct->data[i * k + j];
			sparseStruct->transposed_index[j*n+i]= sparseStruct->col_index[i*k+j];
		}
	}

    //Setting up Kernel variables
    int *d_transposed, *d_transposed_index, *d_vectorx, *d_vectory;
    hipMalloc((void **)&d_transposed,sizeof(int32_t) * sparseStruct->data_len);
    hipMalloc((void **)&d_transposed_index, sizeof(int32_t) * sparseStruct->data_len);
    hipMalloc((void **)&d_vectorx, sizeof(int32_t) * n);
    hipMalloc((void **)&d_vectory, sizeof(int32_t)*n);

	hipMemcpy(d_transposed, sparseStruct->transposed, sizeof(int32_t) * sparseStruct->data_len, hipMemcpyHostToDevice);
	hipMemcpy(d_transposed_index, sparseStruct->transposed_index, sizeof(int32_t) * sparseStruct->data_len, hipMemcpyHostToDevice);
	hipMemcpy(d_vectorx, vectorx, sizeof(int32_t) * n, hipMemcpyHostToDevice);
	hipMemcpy(d_vectory, vectory, sizeof(int32_t) * n, hipMemcpyHostToDevice);

    hipEvent_t tick, tock;
	hipEventCreate(&tick);
	hipEventCreate(&tock);

	hipEventRecord(tick, 0);
	ell_spmv<<<gridDim, blockDim>>>(d_transposed, d_transposed_index, d_vectorx, d_vectory, n, sparseStruct->row_len);
	hipEventRecord(tock, 0);
	hipEventSynchronize(tock);
	
	float time;
	hipEventElapsedTime(&time, tick, tock);
	hipMemcpy(vectory, d_vectory, sizeof(int32_t) * n, hipMemcpyDeviceToHost);

	int *multVect = (int *)calloc(n, sizeof(int));
	char multBuff[50];
	snprintf(multBuff, 50, "Axy_%d.dat", n);
	FILE *multFile = fopen(multBuff, "rb");
	count = fread(multVect, sizeof(int), n, multFile);

	//Checking if final answer is correct
	for (int j = 0; j < n; j++)
	{
		if (multVect[j]!=vectory[j]){
			printf("Error!");
			exit(0);
		}
	}
	printf("Test pass\n");
	printf("Time taken= %f ms\n", time);

	free(fullMatrix);
	free(vectorx);
	free(vectory);
	free(sparseStruct->data);
	free(sparseStruct->col_index);
	free(sparseStruct->transposed_index);
	free(sparseStruct->transposed);
	free(sparseStruct);
	free(multVect);
	fclose(Afile);
	fclose(xfile);
	fclose(yfile);
	fclose(multFile);
	hipFree(d_transposed);
	hipFree(d_transposed_index);
	hipFree(d_vectory);
	hipFree(d_vectorx);

}